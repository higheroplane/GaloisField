#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/for_each.h>
#include <thrust/generate.h>
#include <thrust/copy.h>

#include <thrust/complex.h>
#include <hipfft/hipfft.h>
#include<vector>
#include <algorithm>

using std::vector;


/*void finalize(vector <int> num)
{
    for (int i = 0; i < num.size(); i ++)
    {

        num[i + 1] += num[i] / 10;
        if (num[i] >= 0) num[i] %= 10;
        else {num[i] = 10 + num[i]%10; num[i+1] --;}
    }
}*/

__global__ void finalize(int* num, int size)
{
    for (int i = 0; i < size; i ++)
    {

        num[i + 1] += num[i] / 10;
        if (num[i] >= 0) num[i] %= 10;
        else {num[i] = 10 + num[i]%10; num[i+1] --;}
    }
}

vector<int> naive_mul(const vector<int>& x, const vector<int>& y)
{
    int len = 0;
    if (x.size() > y.size()) len = x.size();
    else len = y.size();
    vector<int> res(2 * len);

    for (int i = 0; i < x.size(); i ++)
        for (int j = 0; j < y.size(); j ++) res[i + j] += x[i] * y[j];

    return res;
}

vector<int> karatsuba_mul(const vector<int>& _x, const vector<int>& _y)
{
//printf ("karr\n");
    vector <int> x(_x), y(_y);
    int len = 0;
    if (x.size() > y.size()) {len = x.size(); y.reserve(len); y.insert(y.end(), x.size() - y.size(), 0);}
    else {len  = y.size(); x.reserve(len); x.insert(x.end(), y.size() - x.size(), 0);}

    x.reserve (len);
    

    vector <int> res(2 * len);

    if (len <= 50) {
        return naive_mul(x, y);
    }

    int k = len / 2, l = len/2 + len%2;
    vector<int> Xr {x.begin(), x.begin() + k};
    vector<int> Xl {x.begin() + k, x.end()};
    vector<int> Yr {y.begin(), y.begin() + k};
    vector<int> Yl {y.begin() + k, y.end()};

    vector<int> P1 = karatsuba_mul(Xl, Yl);
    vector<int> P2 = karatsuba_mul(Xr, Yr);


    vector<int> Xlr(Xl);
    vector<int> Ylr(Yl);

    for (int i = 0; i < k; i ++)
    {
        Xlr[i] += Xr[i];
        Ylr[i] += Yr[i];
    }


    vector<int> P3 = karatsuba_mul(Xlr, Ylr);

    for (int i = 0;   i < P1.size(); i ++) P3 [i] -= P1[i];
    for (int i = 0;   i < P2.size(); i ++) P3 [i] -= P2[i];

    for (int i = 0;   i < P2.size()      ; i ++) res[i]  = P2[i];
    for (int i = 2*k; i < P1.size() + 2*k; i ++) res[i]  = P1[i - 2*k];
    for (int i = k;   i < P3.size() + k  ; i ++) res[i] += P3[i - k];


    return res;
}

struct PrintIntVector
{
  __device__
  void operator () (int val)
  {     
	printf ("%d ", val);
    //printf ("\n");
  }
};


struct PrintComplexVector
{
  __device__
  void operator () (thrust::complex<float> val)
  {     
	printf ("%.1f + %.1fi ", val.real(), val.imag());
    //printf ("\n");
  }
};

struct PrintFloatVector
{
  __device__
  void operator () (float val)
  {     
	printf ("%.1f ", val);
    //printf ("\n");
  }
};

struct GenRand
{
    __device__
    int operator () (int idx)
    {
        thrust::minstd_rand rng;
        thrust::uniform_int_distribution<int> dist(0, 9);
        rng.discard(idx+100);
       // printf ("=%d", dist(rng));
        return (dist(rng) + idx)%10;
    }
};

struct Round
{
    __device__
    int operator () (float val)
    {
        return round(val);
    }
};

const int LEN_  = 2, MAX_ = 50000, STEP_ = 10;


int main()
{
    FILE * f = fopen ("time.txt", "w");
    //FILE * f2 = fopen ("kar.txt", "w");

    thrust::device_vector<int>                    a_int(LEN_, 1), b_int(LEN_, 1);
    thrust::device_vector<float> a_c  (2*LEN_), b_c  (2*LEN_);
    thrust::device_vector<thrust::complex<float>> c_c  (LEN_);
    thrust::device_vector<float> c_f  (LEN_);
    thrust::device_vector<int>                    c_int(LEN_);
    
    vector <int> a_k (LEN_/2), b_k (LEN_/2);

    a_k.reserve(MAX_/2); b_k.reserve(MAX_/2); a_int.reserve(MAX_); b_int.reserve(MAX_); a_c.reserve(MAX_*2); b_c.reserve(MAX_*2);
    c_c.reserve(MAX_); c_int.reserve(MAX_); c_f.reserve(MAX_);
hipfftHandle plan, plan2;
    for (int i = LEN_; i < MAX_; i += STEP_)
    {
        //thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(i/2), a_int.begin(), GenRand());
        //thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(i/2), b_int.begin(), GenRand());
        
        thrust::copy (a_int.begin(), a_int.end(), a_c.begin());
        thrust::copy (b_int.begin(), b_int.end(), b_c.begin());

        int time1 = clock();
        hipfftPlan1d(&plan,  i, HIPFFT_R2C, 1);
        hipfftPlan1d(&plan2, i, HIPFFT_C2R, 1);
	    hipfftExecR2C(plan, (hipfftReal*) a_c.data().get(), (hipfftComplex*) a_c.data().get());	
	    hipfftExecR2C(plan, (hipfftReal*) b_c.data().get(), (hipfftComplex*) b_c.data().get());    
        thrust::transform(thrust::device, (thrust::complex<float>*) a_c.data().get(), (thrust::complex<float>*) a_c.data().get() + i, 
                                         (thrust::complex<float>*) b_c.data().get(), c_c.begin(),  thrust::multiplies<thrust::complex<float>>());
        hipfftExecC2R(plan2, (hipfftComplex *)c_c.data().get(), (hipfftReal *)c_f.data().get());

        thrust::transform (c_f.begin(), c_f.end(), c_int.begin(), [i] __device__ (float val) {return round(val)/i;});
        finalize<<<1,1>>>(c_int.data().get(), c_int.size());
        hipDeviceSynchronize();
        time1 = clock() - time1;
        hipfftDestroy(plan); hipfftDestroy(plan2);
        generate (a_k.begin(), a_k.end(), rand);
        generate (b_k.begin(), b_k.end(), rand);
        int time2 = clock();
        c_int == karatsuba_mul (a_k, b_k);
        finalize<<<1,1>>>(c_int.data().get(), c_int.size());
        time2 = clock() - time2;


        a_k  .insert(a_k  .end(), STEP_/2, 0);
        b_k  .insert(b_k  .end(), STEP_/2, 0);
        a_c  .insert(a_c  .end(), STEP_*2, 0.0);
        b_c  .insert(b_c  .end(), STEP_*2, 0.0);
        a_int.insert(a_int.end(), STEP_,   1);
        b_int.insert(b_int.end(), STEP_,   1);
        c_c  .insert(c_c  .end(), STEP_,   (0.0, 0.0));
        c_f  .insert(c_f  .end(), STEP_,   0.0);
        c_int.insert(c_int.end(), STEP_,   0);
        
        fprintf (f, "%d %d %d\n", i, time1, time2);
        if (i%1000 == 2) printf ("sizea = %d sizeb = %d %d\n", a_c.size(), b_c.size(), i);
    }
    


    /*for (int i = 12; i < 13; i +=2)
    {
       // printf ("started %d it\n", i);        
        thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(i/2), a.begin(), GenRand());
        thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(i/2), b.begin(), GenRand());

        thrust::copy (a.begin(), a.end(), cf.begin());
        thrust::copy (a.begin(), a.end(), cf.begin());
       // printf ("arrays prepared\n");
        
        ////////////////////////
	    hipEvent_t start, stop;
	    float time;
	    hipEventCreate(&start);
	    hipEventRecord(start, 0);
	    hipEventCreate(&stop);
        ////////////////////////
	    int time1 = clock();
        hipfftPlan1d(&plan, i, HIPFFT_R2C, 1);
	    hipfftExecR2C(plan, (hipfftReal *)af.data().get(), (hipfftComplex *)af.data().get());	
	    hipfftExecR2C(plan, (hipfftReal *)bf.data().get(), (hipfftComplex *)bf.data().get());
        thrust::transform(af.begin(), af.end(), bf.begin(), cf.begin(),  thrust::multiplies<thrust::complex<float>>());
        hipfftExecC2R(plan, (hipfftComplex *)cf.data().get(), (hipfftReal *)cf.data().get());
        hipDeviceSynchronize();
        time1 = clock() - time1;
        //////////////////////////////////////////////
	    hipEventRecord(stop, 0);
	    hipEventSynchronize(stop);
	    hipEventElapsedTime(&time, start, stop);
	 //   printf ("Time for the kernel: %f ms\n", time);
	    //////////////////////////////////////////////
srand(13);

        int time2 = clock();
   // printf("haha\n");
        vector <int> ak (i);
        vector <int> bk (i);
        generate (ak.begin(), ak.end(), rand);
        generate (bk.begin(), bk.end(), rand);
        c = karatsuba_mul (ak, bk);
        time2 = clock() - time2;

        fprintf (f, "%d %d %d\n", i, time1, time*1000);
        //printf ("%d %f %f\n", i, time*1000.0, ((float)time2)/CLOCKS_PER_SEC);
        a.push_back(0);
        b.push_back(0);
        af.push_back(0.0);
        bf.push_back(0.0);
        a.push_back(0);
        b.push_back(0);
        af.push_back(0.0);
        bf.push_back(0.0);
    }*/
     
    //hipDeviceSynchronize();
    //thrust::for_each (arr.begin(), arr.begin() + 5, PrintVector()); 
    
    return 0;
}

